
#include <hip/hip_runtime.h>
#include <iostream>
#include <getopt.h>

#define no_argument 0
#define required_argument 1 
#define optional_argument 2


int main(int argc, char * argv[])
{
  std::cout << "Hello" << std::endl;

  const struct option long_opts[] =
  {
    {"version",     no_argument,        0, 'v'},
    {"help",        no_argument,        0, 'h'},
    {"metric_set",  required_argument,  0, 'm'},
    {"eval",        required_argument,  0, 'e'},
    {0,0,0,0},
  };

  int index;
  int iarg=0;

  //turn off getopt error message
  opterr=1; 

  while(iarg != -1)
  {
    iarg = getopt_long_only(argc, argv, "e:m:h", long_opts, &index);

    switch (iarg)
    {
      case 'e':
        std::cout << "You hit eval" << std::endl;
        break;

      case 'm':
        std::cout << "You hit metric_set: " << optarg << std::endl;
        break;

      case 'h':
        std::cout << "You hit help" << std::endl;
        break;
    }
  }

  std::cout << "GoodBye!" << std::endl;

  return 0; 
}