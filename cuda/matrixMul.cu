// Multiply two matrices A * B = C

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "matrixMul_kernel.cu"

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
   for (int i = 0; i < size; ++i)
   data[i] = rand() / (float)RAND_MAX;
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////

int main(int argc, char** argv) {
   // set seed for rand()
   srand(2006);

   // 1. allocate host memory for matrices A and B
   unsigned int size_A = WA * HA;
   unsigned int mem_size_A = sizeof(float) * size_A;
   float* h_A = (float*) malloc(mem_size_A);

   unsigned int size_B = WB * HB;
   unsigned int mem_size_B = sizeof(float) * size_B;
   float* h_B = (float*) malloc(mem_size_B);

   // 2. initialize host memory
   randomInit(h_A, size_A);
   randomInit(h_B, size_B);

   // 8. allocate device memory
   float* d_A;
   float* d_B;
   hipMalloc((void**) &d_A, mem_size_A);
   hipMalloc((void**) &d_B, mem_size_B);

   // 9. copy host memory to device
   hipMemcpy(d_A, h_A, mem_size_A,
   hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, mem_size_B,
   hipMemcpyHostToDevice);

   // 4. allocate host memory for the result C
   unsigned int size_C = WC * HC;
   unsigned int mem_size_C = sizeof(float) * size_C;
   float* h_C = (float*) malloc(mem_size_C);

   // 10. allocate device memory for the result
   float* d_C;
   hipMalloc((void**) &d_C, mem_size_C);

   // 5. perform the calculation
   // setup execution parameters
   dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
   dim3 grid(WC / threads.x, HC / threads.y);

   // execute the kernel
   matrixMul<<< grid, threads >>>(d_C, d_A,
                                  d_B, WA, WB);

   // 11. copy result from device to host
   hipMemcpy(h_C, d_C, mem_size_C,
   hipMemcpyDeviceToHost);

   // 7. clean up memory
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}


   int devID;
   hipDeviceProp_t props;
   checkCudaErrors(hipGetDevice(&devID));
   checkCudaErrors(hipGetDeviceProperties(&props, devID));

   int block_size = (props.major < 2) ? 16 : 32;
   unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
   uiWA = uiHA= uiWB = uiHB = uiWC = uiHC;

   // allocate host memory for matrices A and B
   unsigned int size_A = uiWA * uiHA;
   unsigned int mem_size_A = sizeof(float) * size_A;
   float* h_A = (float*)malloc(mem_size_A);
   unsigned int size_B = uiWB * uiHB;
   unsigned int mem_size_B = sizeof(float) * size_B;
   float* h_B = (float*)malloc(mem_size_B);

   // initialize host memory
   srand(2012);
   randomInit(h_A, size_A);
   randomInit(h_B, size_B);

   // allocate device memory
   float* d_A, *d_B, *d_C;
   unsigned int size_C = uiWC * uiHC;
   unsigned int mem_size_C = sizeof(float) * size_C;

   // allocate host memory for the result
   float* h_C      = (float*) malloc(mem_size_C);
   float* h_CUBLAS = (float*) malloc(mem_size_C);
   checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
   checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));
   // copy host memory to device
   checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
   checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
   checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));
   // setup execution parameters
   dim3 threads(block_size, block_size);
   dim3 grid(uiWC / threads.x, uiHC / threads.y);

   //Performs warmup operation using matrixMul CUDA kernel
   if (block_size 16) {
   matrixMul<16><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
   } else {
   matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
   }

   hipDeviceSynchronize();

   // clean up memory
   free(h_A);
   free(h_B);
   free(h_C);