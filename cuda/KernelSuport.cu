#ifndef _KERNEL_SUPPORT_
#define _KERNEL_SUPPORT_

#include <iostream>
#include "MyKernel.h"

int main( int argc, char** argv)
{
    const int N = 16;

    int hostArray[N];
    int *deviceArray;

    std::cout << "Before device\n";
    for(int i = 0; i < N; i++) {
        hostArray[i] = i + 1;
        std::cout << hostArray[i] << "\n";
    }
    std::cout << "\n";

    hipMalloc((void**) &deviceArray, N*sizeof(int));

    hipMemcpy(deviceArray, hostArray, N*sizeof(int), hipMemcpyHostToDevice);
    TestDevice <<< 4, 4 >>> (deviceArray);
    hipMemcpy(hostArray, deviceArray, N*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceArray);

    std::cout << "After device\n";
    for(int i = 0; i < N; i++) std::cout << hostArray[i] << "\n";

    std::cout << "Done\n";
}

#endif