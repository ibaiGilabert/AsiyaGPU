 
#include <hip/hip_runtime.h>
#include "stdio.h"


__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main() {
	int a = 3;
	int b = 4;
	int c, *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));
	add<<<1,1>>>(a, b, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c);

	printf("%d + %d = %d\n", a, b, c);
}