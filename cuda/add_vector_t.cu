#include "hip/hip_runtime.h"
#include "stdio.h"

#include <iostream>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

using namespace std;

#define N 10


/*__global__ void add(int &a, int &b, int &c) {
	int tId = blockIdx.x*blockDim.x + threadIdx.x;
	if (tId < N)
		c[tId] = a[tId] + b[tId];
}*/

int main() {
	thrust::host_vector<float> A(N);
	thrust::host_vector<float> B(N);
	for(int i = 0; i < N; ++i) {
		A[i] = i;
		B[i] = i;
	}
	thrust::device_vector<float> dA = A;
	thrust::device_vector<float> dB = B;
	thrust::device_vector<float> C(N);

	thrust::transform(dA.begin(), dA.end()),
					  dB.begin(),
					  C.begin(),
					  thrust::plus<float>());
}