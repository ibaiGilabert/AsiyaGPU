#include "hip/hip_runtime.h"
#include "MyKernel.h"

__global__ void TestDevice(int *deviceArray)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    deviceArray[idx] = deviceArray[idx]*deviceArray[idx];
}
